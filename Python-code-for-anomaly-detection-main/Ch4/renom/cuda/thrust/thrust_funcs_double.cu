#include "hip/hip_runtime.h"
#include "thrust_funcs_double.h"


__device__ double renom_atomicAdd(double* address, double val)
{
    unsigned long long int* address_as_ull = (unsigned long long int*)address;

    unsigned long long int old = *address_as_ull, assumed;

    do{ assumed = old;
        old = atomicCAS(address_as_ull, assumed,__double_as_longlong(val +__longlong_as_double(assumed)));
    } while (assumed != old);

    return __longlong_as_double(old);
}

#define USE_RENOM_ATOMICADD

#include "thrust_funcs.inl"
